#include "dma_transfer.h"
#include <stdexcept>
#include <iostream>  // 添加这行


DMATransfer::DMATransfer() {
    CUDA_CHECK(hipStreamCreate(&stream_));
    setup_timing_events();
}

DMATransfer::~DMATransfer() {
    hipEventDestroy(start_event_);
    hipEventDestroy(stop_event_);
    hipStreamDestroy(stream_);
}

void DMATransfer::setup_timing_events() {
    CUDA_CHECK(hipEventCreate(&start_event_));
    CUDA_CHECK(hipEventCreate(&stop_event_));
}

double DMATransfer::get_elapsed_time() {
    float milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start_event_, stop_event_));
    return static_cast<double>(milliseconds);
}

// H2D DMA传输实现
double DMATransfer::benchmark_h2d_sync_pageable(void* d_dst, void* h_src, size_t size) {
    // 预热
    CUDA_CHECK(hipMemcpy(d_dst, h_src, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipDeviceSynchronize());
    
    // 计时
    CUDA_CHECK(hipEventRecord(start_event_));
    CUDA_CHECK(hipMemcpy(d_dst, h_src, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipEventRecord(stop_event_));
    CUDA_CHECK(hipEventSynchronize(stop_event_));
    
    return get_elapsed_time();
}

double DMATransfer::benchmark_h2d_async_pageable(void* d_dst, void* h_src, size_t size) {
    // 预热
    CUDA_CHECK(hipMemcpyAsync(d_dst, h_src, size, hipMemcpyHostToDevice, stream_));
    CUDA_CHECK(hipStreamSynchronize(stream_));
    
    // 计时
    CUDA_CHECK(hipEventRecord(start_event_, stream_));
    CUDA_CHECK(hipMemcpyAsync(d_dst, h_src, size, hipMemcpyHostToDevice, stream_));
    CUDA_CHECK(hipEventRecord(stop_event_, stream_));
    CUDA_CHECK(hipStreamSynchronize(stream_));
    
    return get_elapsed_time();
}

double DMATransfer::benchmark_h2d_sync_pinned(void* d_dst, void* h_pinned_src, size_t size) {
    // 预热
    CUDA_CHECK(hipMemcpy(d_dst, h_pinned_src, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipDeviceSynchronize());
    
    // 计时
    CUDA_CHECK(hipEventRecord(start_event_));
    CUDA_CHECK(hipMemcpy(d_dst, h_pinned_src, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipEventRecord(stop_event_));
    CUDA_CHECK(hipEventSynchronize(stop_event_));
    
    return get_elapsed_time();
}

double DMATransfer::benchmark_h2d_async_pinned(void* d_dst, void* h_pinned_src, size_t size) {
    // 预热
    CUDA_CHECK(hipMemcpyAsync(d_dst, h_pinned_src, size, hipMemcpyHostToDevice, stream_));
    CUDA_CHECK(hipStreamSynchronize(stream_));
    
    // 计时
    CUDA_CHECK(hipEventRecord(start_event_, stream_));
    CUDA_CHECK(hipMemcpyAsync(d_dst, h_pinned_src, size, hipMemcpyHostToDevice, stream_));
    CUDA_CHECK(hipEventRecord(stop_event_, stream_));
    CUDA_CHECK(hipStreamSynchronize(stream_));
    
    return get_elapsed_time();
}

double DMATransfer::benchmark_h2d_mapped(void* d_dst, void* h_mapped_src, size_t size) {
    // 对于mapped memory，实际上是zero-copy，不需要DMA传输
    // 但这里我们测试显式拷贝的性能
    CUDA_CHECK(hipEventRecord(start_event_, stream_));
    CUDA_CHECK(hipMemcpyAsync(d_dst, h_mapped_src, size, hipMemcpyHostToDevice, stream_));
    CUDA_CHECK(hipEventRecord(stop_event_, stream_));
    CUDA_CHECK(hipStreamSynchronize(stream_));
    
    return get_elapsed_time();
}

// D2H DMA传输实现
double DMATransfer::benchmark_d2h_sync_pageable(void* h_dst, void* d_src, size_t size) {
    // 预热
    CUDA_CHECK(hipMemcpy(h_dst, d_src, size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize());
    
    // 计时
    CUDA_CHECK(hipEventRecord(start_event_));
    CUDA_CHECK(hipMemcpy(h_dst, d_src, size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipEventRecord(stop_event_));
    CUDA_CHECK(hipEventSynchronize(stop_event_));
    
    return get_elapsed_time();
}

double DMATransfer::benchmark_d2h_async_pageable(void* h_dst, void* d_src, size_t size) {
    // 预热
    CUDA_CHECK(hipMemcpyAsync(h_dst, d_src, size, hipMemcpyDeviceToHost, stream_));
    CUDA_CHECK(hipStreamSynchronize(stream_));
    
    // 计时
    CUDA_CHECK(hipEventRecord(start_event_, stream_));
    CUDA_CHECK(hipMemcpyAsync(h_dst, d_src, size, hipMemcpyDeviceToHost, stream_));
    CUDA_CHECK(hipEventRecord(stop_event_, stream_));
    CUDA_CHECK(hipStreamSynchronize(stream_));
    
    return get_elapsed_time();
}

double DMATransfer::benchmark_d2h_sync_pinned(void* h_pinned_dst, void* d_src, size_t size) {
    // 预热
    CUDA_CHECK(hipMemcpy(h_pinned_dst, d_src, size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize());
    
    // 计时
    CUDA_CHECK(hipEventRecord(start_event_));
    CUDA_CHECK(hipMemcpy(h_pinned_dst, d_src, size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipEventRecord(stop_event_));
    CUDA_CHECK(hipEventSynchronize(stop_event_));
    
    return get_elapsed_time();
}

double DMATransfer::benchmark_d2h_async_pinned(void* h_pinned_dst, void* d_src, size_t size) {
    // 预热
    CUDA_CHECK(hipMemcpyAsync(h_pinned_dst, d_src, size, hipMemcpyDeviceToHost, stream_));
    CUDA_CHECK(hipStreamSynchronize(stream_));
    
    // 计时
    CUDA_CHECK(hipEventRecord(start_event_, stream_));
    CUDA_CHECK(hipMemcpyAsync(h_pinned_dst, d_src, size, hipMemcpyDeviceToHost, stream_));
    CUDA_CHECK(hipEventRecord(stop_event_, stream_));
    CUDA_CHECK(hipStreamSynchronize(stream_));
    
    return get_elapsed_time();
}

double DMATransfer::benchmark_d2h_mapped(void* h_mapped_dst, void* d_src, size_t size) {
    CUDA_CHECK(hipEventRecord(start_event_, stream_));
    CUDA_CHECK(hipMemcpyAsync(h_mapped_dst, d_src, size, hipMemcpyDeviceToHost, stream_));
    CUDA_CHECK(hipEventRecord(stop_event_, stream_));
    CUDA_CHECK(hipStreamSynchronize(stream_));
    
    return get_elapsed_time();
}

// D2D DMA传输实现
double DMATransfer::benchmark_d2d_sync(void* d_dst, void* d_src, size_t size) {
    // 预热
    CUDA_CHECK(hipMemcpy(d_dst, d_src, size, hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipDeviceSynchronize());
    
    // 计时
    CUDA_CHECK(hipEventRecord(start_event_));
    CUDA_CHECK(hipMemcpy(d_dst, d_src, size, hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipEventRecord(stop_event_));
    CUDA_CHECK(hipEventSynchronize(stop_event_));
    
    return get_elapsed_time();
}

double DMATransfer::benchmark_d2d_async(void* d_dst, void* d_src, size_t size) {
    // 预热
    CUDA_CHECK(hipMemcpyAsync(d_dst, d_src, size, hipMemcpyDeviceToDevice, stream_));
    CUDA_CHECK(hipStreamSynchronize(stream_));
    
    // 计时
    CUDA_CHECK(hipEventRecord(start_event_, stream_));
    CUDA_CHECK(hipMemcpyAsync(d_dst, d_src, size, hipMemcpyDeviceToDevice, stream_));
    CUDA_CHECK(hipEventRecord(stop_event_, stream_));
    CUDA_CHECK(hipStreamSynchronize(stream_));
    
    return get_elapsed_time();
}

// 修复DMA P2P同步传输
double DMATransfer::benchmark_p2p_sync(void* d_dst, void* d_src, size_t size, int dst_device, int src_device) {
    int original_device;
    CUDA_CHECK(hipGetDevice(&original_device));
    
    try {
        // 检查P2P访问能力
        int can_access_peer;
        CUDA_CHECK(hipDeviceCanAccessPeer(&can_access_peer, dst_device, src_device));
        if (!can_access_peer) {
            std::cout << "P2P access not supported between devices " 
                      << src_device << " and " << dst_device << std::endl;
            CUDA_CHECK(hipSetDevice(original_device));
            return 0.0;
        }
        
        // 启用P2P访问（双向）
        CUDA_CHECK(hipSetDevice(dst_device));
        hipError_t p2p_result = hipDeviceEnablePeerAccess(src_device, 0);
        if (p2p_result != hipSuccess && p2p_result != hipErrorPeerAccessAlreadyEnabled) {
            CUDA_CHECK(hipSetDevice(original_device));
            return 0.0;
        }
        
        CUDA_CHECK(hipSetDevice(src_device));
        p2p_result = hipDeviceEnablePeerAccess(dst_device, 0);
        if (p2p_result != hipSuccess && p2p_result != hipErrorPeerAccessAlreadyEnabled) {
            CUDA_CHECK(hipSetDevice(original_device));
            return 0.0;
        }
        
        // 使用目标设备执行P2P传输
        CUDA_CHECK(hipSetDevice(dst_device));
        
        // 预热
        CUDA_CHECK(hipMemcpy(d_dst, d_src, size, hipMemcpyDeviceToDevice));
        CUDA_CHECK(hipDeviceSynchronize());
        
        // 创建专用的事件用于计时
        hipEvent_t p2p_start, p2p_stop;
        CUDA_CHECK(hipEventCreate(&p2p_start));
        CUDA_CHECK(hipEventCreate(&p2p_stop));
        
        // 计时
        CUDA_CHECK(hipEventRecord(p2p_start));
        CUDA_CHECK(hipMemcpy(d_dst, d_src, size, hipMemcpyDeviceToDevice));
        CUDA_CHECK(hipEventRecord(p2p_stop));
        CUDA_CHECK(hipEventSynchronize(p2p_stop));
        
        float milliseconds = 0;
        CUDA_CHECK(hipEventElapsedTime(&milliseconds, p2p_start, p2p_stop));
        
        // 清理
        CUDA_CHECK(hipEventDestroy(p2p_start));
        CUDA_CHECK(hipEventDestroy(p2p_stop));
        CUDA_CHECK(hipSetDevice(original_device));
        
        return static_cast<double>(milliseconds);
        
    } catch (const std::exception& e) {
        CUDA_CHECK(hipSetDevice(original_device));
        std::cerr << "P2P sync transfer error: " << e.what() << std::endl;
        return 0.0;
    }
}

// 修复DMA P2P异步传输
double DMATransfer::benchmark_p2p_async(void* d_dst, void* d_src, size_t size, int dst_device, int src_device) {
    int original_device;
    CUDA_CHECK(hipGetDevice(&original_device));
    
    try {
        // 检查P2P访问能力
        int can_access_peer;
        CUDA_CHECK(hipDeviceCanAccessPeer(&can_access_peer, dst_device, src_device));
        if (!can_access_peer) {
            CUDA_CHECK(hipSetDevice(original_device));
            return 0.0;
        }
        
        // 启用P2P访问
        CUDA_CHECK(hipSetDevice(dst_device));
        hipError_t p2p_result = hipDeviceEnablePeerAccess(src_device, 0);
        if (p2p_result != hipSuccess && p2p_result != hipErrorPeerAccessAlreadyEnabled) {
            CUDA_CHECK(hipSetDevice(original_device));
            return 0.0;
        }
        
        CUDA_CHECK(hipSetDevice(src_device));
        p2p_result = hipDeviceEnablePeerAccess(dst_device, 0);
        if (p2p_result != hipSuccess && p2p_result != hipErrorPeerAccessAlreadyEnabled) {
            CUDA_CHECK(hipSetDevice(original_device));
            return 0.0;
        }
        
        // 在目标设备上创建流和事件
        CUDA_CHECK(hipSetDevice(dst_device));
        
        hipStream_t p2p_stream;
        hipEvent_t p2p_start, p2p_stop;
        CUDA_CHECK(hipStreamCreate(&p2p_stream));
        CUDA_CHECK(hipEventCreate(&p2p_start));
        CUDA_CHECK(hipEventCreate(&p2p_stop));
        
        // 预热
        CUDA_CHECK(hipMemcpyAsync(d_dst, d_src, size, hipMemcpyDeviceToDevice, p2p_stream));
        CUDA_CHECK(hipStreamSynchronize(p2p_stream));
        
        // 计时
        CUDA_CHECK(hipEventRecord(p2p_start, p2p_stream));
        CUDA_CHECK(hipMemcpyAsync(d_dst, d_src, size, hipMemcpyDeviceToDevice, p2p_stream));
        CUDA_CHECK(hipEventRecord(p2p_stop, p2p_stream));
        CUDA_CHECK(hipStreamSynchronize(p2p_stream));
        
        float milliseconds = 0;
        CUDA_CHECK(hipEventElapsedTime(&milliseconds, p2p_start, p2p_stop));
        
        // 清理
        CUDA_CHECK(hipEventDestroy(p2p_start));
        CUDA_CHECK(hipEventDestroy(p2p_stop));
        CUDA_CHECK(hipStreamDestroy(p2p_stream));
        CUDA_CHECK(hipSetDevice(original_device));
        
        return static_cast<double>(milliseconds);
        
    } catch (const std::exception& e) {
        CUDA_CHECK(hipSetDevice(original_device));
        std::cerr << "P2P async transfer error: " << e.what() << std::endl;
        return 0.0;
    }
}